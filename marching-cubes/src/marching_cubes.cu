#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>

#include <array>
#include <geometry_msgs/msg/point.hpp>
#include <geometry_msgs/msg/transform_stamped.hpp>
#include <rclcpp/rclcpp.hpp>
#include <tf2_geometry_msgs/tf2_geometry_msgs.hpp>
#include <visualization_msgs/msg/marker.hpp>
#include <visualization_msgs/msg/marker_array.hpp>

#include "tf2_ros/buffer.h"
#include "tf2_ros/transform_listener.h"

template <typename T>
struct Vec3 {
  T x, y, z;
  Vec3(const T x, const T y, const T z) : x(x), y(y), z(z){};
  uint32_t total() const { return x * y * z; }

  Vec3<T> operator*(const T scale) {
    Vec3<T> result(*this);
    result->x *= scale;
    result->y *= scale;
    result->z *= scale;
    return result;
  }

  Vec3<T> operator+(const Vec3<T>& v2) {
    Vec3<T> result(*this);
    result.x += v2.x;
    result.y += v2.y;
    result.z += v2.z;
    return result;
  };
};

__global__ void d_updateHeights(const float time, float* d_heights) {
  const uint32_t i = blockIdx.x;
  const uint32_t j = threadIdx.x;
  const uint32_t idx = i * blockDim.x + j;
  if (i < 50 || i > (gridDim.x - 50) || j < 50 || j > (blockDim.x - 50))
    d_heights[idx] = 0.01;
  else
    d_heights[idx] = 0.5 + 0.2 * sin(time + 2.0 * 3.14 * float(i) / gridDim.x) * cos(2.0 * 3.14 * float(j) / blockDim.x);
}

__global__ void d_updateVolumes(const uint8_t* d_groundCubes, const uint8_t* d_bucketCubes, const float* d_sampleVolumes, float* d_groundVolumes,
                                float* d_bucketVolumes) {
  const uint32_t idx = blockIdx.x * gridDim.y * blockDim.x + blockIdx.y * blockDim.x + threadIdx.x;
  d_groundVolumes[idx] = d_sampleVolumes[d_groundCubes[idx]];
  d_bucketVolumes[idx] = d_sampleVolumes[d_bucketCubes[idx]];
}

__global__ void d_updateNodes(const float* d_heights, const float size, bool* d_groundNodes, bool* d_bucketNodes, const float min_x,
                              const float max_x, const float min_y, const float max_y, const float min_z, const float max_z) {
  const uint32_t idx = blockIdx.x * gridDim.y * blockDim.x + blockIdx.y * blockDim.x + threadIdx.x;
  const uint32_t h_idx = blockIdx.x * gridDim.y + blockIdx.y;
  float z = threadIdx.x * size;
  if (!d_bucketNodes[idx]) d_groundNodes[idx] = d_heights[h_idx] >= z;

  if (d_groundNodes[idx]) {
    float p_z = threadIdx.x * size;
    float p_y = blockIdx.y * size;
    float p_x = blockIdx.x * size;
    if (p_x > min_x && p_x < max_x && p_y > min_y && p_y < max_y && p_z > min_z && p_z < max_z) {
      d_bucketNodes[idx] = 1;
      d_groundNodes[idx] = 0;
      // d_bucketNodes[idx] = !d_groundNodes[idx]};
    }
  }
};

__global__ void d_updateCubes(const bool* d_groundNodes, const bool* d_bucketNodes, uint8_t* d_groundCubes, uint8_t* d_bucketCubes) {
  const uint32_t i = blockIdx.x;
  const uint32_t j = blockIdx.y;
  const uint32_t k = threadIdx.x;

  const uint32_t idx = i * gridDim.y * blockDim.x + j * blockDim.x + k;

  const uint32_t n_idx[] = {(i + 0) * (gridDim.y + 1) * (blockDim.x + 1) + (j + 0) * (blockDim.x + 1) + (k + 0),
                            (i + 1) * (gridDim.y + 1) * (blockDim.x + 1) + (j + 0) * (blockDim.x + 1) + (k + 0),
                            (i + 1) * (gridDim.y + 1) * (blockDim.x + 1) + (j + 1) * (blockDim.x + 1) + (k + 0),
                            (i + 0) * (gridDim.y + 1) * (blockDim.x + 1) + (j + 1) * (blockDim.x + 1) + (k + 0),
                            (i + 0) * (gridDim.y + 1) * (blockDim.x + 1) + (j + 0) * (blockDim.x + 1) + (k + 1),
                            (i + 1) * (gridDim.y + 1) * (blockDim.x + 1) + (j + 0) * (blockDim.x + 1) + (k + 1),
                            (i + 1) * (gridDim.y + 1) * (blockDim.x + 1) + (j + 1) * (blockDim.x + 1) + (k + 1),
                            (i + 0) * (gridDim.y + 1) * (blockDim.x + 1) + (j + 1) * (blockDim.x + 1) + (k + 1)};

  uint8_t res = 0;
  for (int8_t i = 7; i >= 0; --i) res += (d_groundNodes[n_idx[i]] << i);
  d_groundCubes[idx] = res;
  res = 0;
  for (int8_t i = 7; i >= 0; --i) res += (d_bucketNodes[n_idx[i]] << i);
  d_bucketCubes[idx] = res;
};

using namespace std::chrono_literals;

class MarchingCubes : public rclcpp::Node {
 public:
  MarchingCubes(const uint32_t x_count, const uint32_t y_count, const uint32_t z_count, const float size) : Node("marching_cube_node") {
    m_tfBuffer = std::make_unique<tf2_ros::Buffer>(this->get_clock());
    m_tfListener = std::make_shared<tf2_ros::TransformListener>(*m_tfBuffer);

    // m_Timer = this->create_wall_timer(200ms, std::bind(&MarchingCubes::callback, this));
    m_markerPublisher = this->create_publisher<visualization_msgs::msg::MarkerArray>("marchingCubes", 10);
    m_bucketSubsciber = this->create_subscription<visualization_msgs::msg::MarkerArray>(
        "boundingBox/Excavator_bucket", 10, std::bind(&MarchingCubes::callback, this, std::placeholders::_1));
    m_markerArray.markers.resize(3);
    m_markerArray.markers.at(0).ns = "mc";
    m_markerArray.markers.at(0).id = 0;
    m_markerArray.markers.at(0).action = visualization_msgs::msg::Marker::DELETEALL;

    m_markerArray.markers.at(1).ns = "mc_ground";
    m_markerArray.markers.at(1).id = 1;
    m_markerArray.markers.at(1).type = visualization_msgs::msg::Marker::TRIANGLE_LIST;
    m_markerArray.markers.at(1).action = visualization_msgs::msg::Marker::ADD;
    m_markerArray.markers.at(1).header.frame_id = "container";
    m_markerArray.markers.at(1).scale.x = 1.0f;
    m_markerArray.markers.at(1).scale.y = 1.0f;
    m_markerArray.markers.at(1).scale.z = 1.0f;
    m_markerArray.markers.at(1).color.r = 1.0f;
    m_markerArray.markers.at(1).color.g = 0.5f;
    m_markerArray.markers.at(1).color.b = 0.0f;
    m_markerArray.markers.at(1).color.a = 1.0f;
    // m_markerArray.markers.at(1).pose.position.x = 1.0f;
    // m_markerArray.markers.at(1).pose.position.y = -0.5 * size * y_count;
    // m_markerArray.markers.at(1).pose.position.z = -1.0f;
    m_markerArray.markers.at(1).pose.position.x = 0.0f;
    m_markerArray.markers.at(1).pose.position.y = 0.0f;
    m_markerArray.markers.at(1).pose.position.z = 0.0f;

    m_markerArray.markers.at(2).ns = "mc_bucket";
    m_markerArray.markers.at(2).id = 2;
    m_markerArray.markers.at(2).type = visualization_msgs::msg::Marker::TRIANGLE_LIST;
    m_markerArray.markers.at(2).action = visualization_msgs::msg::Marker::ADD;
    m_markerArray.markers.at(2).header.frame_id = "container";
    m_markerArray.markers.at(2).scale.x = 1.0f;
    m_markerArray.markers.at(2).scale.y = 1.0f;
    m_markerArray.markers.at(2).scale.z = 1.0f;
    m_markerArray.markers.at(2).color.r = 0.0f;
    m_markerArray.markers.at(2).color.g = 1.0f;
    m_markerArray.markers.at(2).color.b = 0.0f;
    m_markerArray.markers.at(2).color.a = 1.0f;
    m_markerArray.markers.at(2).pose.position.x = 0.0f;
    m_markerArray.markers.at(2).pose.position.y = 0.0f;
    m_markerArray.markers.at(2).pose.position.z = 0.0f;

    m_size = size;
    m_cubeCount.x = x_count;
    m_cubeCount.y = y_count;
    m_cubeCount.z = z_count;
    m_nodeCount.x = x_count + 1;
    m_nodeCount.y = y_count + 1;
    m_nodeCount.z = z_count + 1;

    d_groundCubes.resize(m_cubeCount.total());
    thrust::fill(d_groundCubes.begin(), d_groundCubes.end(), 0);
    h_groundCubes.resize(m_cubeCount.total());
    thrust::fill(h_groundCubes.begin(), h_groundCubes.end(), 0);
    d_groundVolumes.resize(m_cubeCount.total());
    thrust::fill(d_groundVolumes.begin(), d_groundVolumes.end(), 0.0f);
    d_groundNodes.resize(m_nodeCount.total());
    thrust::fill(d_groundNodes.begin(), d_groundNodes.end(), 0);
    d_groundHeights.resize(m_nodeCount.x * m_nodeCount.y);

    d_bucketCubes.resize(m_cubeCount.total());
    thrust::fill(d_bucketCubes.begin(), d_bucketCubes.end(), 0);
    h_bucketCubes.resize(m_cubeCount.total());
    thrust::fill(h_bucketCubes.begin(), h_bucketCubes.end(), 0);
    d_bucketVolumes.resize(m_cubeCount.total());
    thrust::fill(d_bucketVolumes.begin(), d_bucketVolumes.end(), 0.0f);
    d_bucketNodes.resize(m_nodeCount.total());
    thrust::fill(d_bucketNodes.begin(), d_bucketNodes.end(), 0);

    m_startTime = this->get_clock()->now().seconds();

    hipDeviceSynchronize();
    updateHeight();
    updateMessage();
  };

  void updateHeight() {
    auto time = this->get_clock()->now();
    d_updateHeights<<<m_nodeCount.x, m_nodeCount.y>>>(float(time.seconds() - m_startTime), thrust::raw_pointer_cast(d_groundHeights.data()));
    hipDeviceSynchronize();
  }

  void callback(const visualization_msgs::msg::MarkerArray::SharedPtr msg) {
    if (count >= 0) {
      RCLCPP_INFO(this->get_logger(), "Loading Data : %d %%", 100 - count);
      --count;
      return;
    }
    auto t1 = m_tfBuffer->lookupTransform("container", msg->markers.at(0).header.frame_id, tf2::TimePointZero);
    auto t2 = m_tfBuffer->lookupTransform("container", msg->markers.at(0).header.frame_id, tf2::TimePointZero);
    t2.transform.translation.x = msg->markers.at(0).pose.position.x;
    t2.transform.translation.y = msg->markers.at(0).pose.position.y;
    t2.transform.translation.z = msg->markers.at(0).pose.position.z;
    t2.transform.rotation.x = msg->markers.at(0).pose.orientation.x;
    t2.transform.rotation.y = msg->markers.at(0).pose.orientation.y;
    t2.transform.rotation.z = msg->markers.at(0).pose.orientation.z;
    t2.transform.rotation.w = msg->markers.at(0).pose.orientation.w;

    visualization_msgs::msg::Marker bucketPoints;
    bucketPoints.set__points(msg->markers.at(0).points);

    for (size_t i = 0; i < bucketPoints.points.size(); ++i) {
      tf2::doTransform(msg->markers.at(0).points.at(i), bucketPoints.points.at(i), t2);
      tf2::doTransform(bucketPoints.points.at(i), bucketPoints.points.at(i), t1);
    }

    float x[2] = {1e6, -1e6};
    float y[2] = {1e6, -1e6};
    float z[2] = {1e6, -1e6};
    for (uint8_t i = 0; i < 8; ++i) {
      x[0] = std::min(x[0], float(bucketPoints.points.at(i).x));
      x[1] = std::max(x[1], float(bucketPoints.points.at(i).x));
      y[0] = std::min(y[0], float(bucketPoints.points.at(i).y));
      y[1] = std::max(y[1], float(bucketPoints.points.at(i).y));
      z[0] = std::min(z[0], float(bucketPoints.points.at(i).z));
      z[1] = std::max(z[1], float(bucketPoints.points.at(i).z));
    }
    updateNodes(x[0], x[1], y[0], y[1], z[0], z[1]);
    updateCubes();
    updateVolumes();

    updateMessage();
    // auto start = std::chrono::steady_clock::now();
    // auto end = std::chrono::steady_clock::now();
    // auto dur = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // RCLCPP_INFO(this->get_logger(), "%f took %ld ms, total triangles : %ld", m_totalVolume, dur, m_markerArray.markers.at(1).points.size());
  };

  void updateVolumes() {
    d_updateVolumes<<<dim3(m_cubeCount.x, m_cubeCount.y, 1), m_cubeCount.z>>>(
        thrust::raw_pointer_cast(d_groundCubes.data()), thrust::raw_pointer_cast(d_bucketCubes.data()),
        thrust::raw_pointer_cast(d_sampleVolumes.data()), thrust::raw_pointer_cast(d_groundVolumes.data()),
        thrust::raw_pointer_cast(d_bucketVolumes.data()));
    hipDeviceSynchronize();
    m_totalGroundVolume = thrust::reduce(d_groundVolumes.cbegin(), d_groundVolumes.cend(), 0.0f);
    m_totalBucketVolume = thrust::reduce(d_bucketVolumes.cbegin(), d_bucketVolumes.cend(), 0.0f);
  };

  void updateNodes(const double min_x, const double max_x, const double min_y, const double max_y, const double min_z, const double max_z) {
    d_updateNodes<<<dim3(m_nodeCount.x, m_nodeCount.y, 1), m_nodeCount.z>>>(
        thrust::raw_pointer_cast(d_groundHeights.data()), m_size, thrust::raw_pointer_cast(d_groundNodes.data()),
        thrust::raw_pointer_cast(d_bucketNodes.data()), min_x, max_x, min_y, max_y, min_z, max_z);
    hipDeviceSynchronize();
  };

  void updateCubes() {
    d_updateCubes<<<dim3(m_cubeCount.x, m_cubeCount.y, 1), m_cubeCount.z>>>(
        thrust::raw_pointer_cast(d_groundNodes.data()), thrust::raw_pointer_cast(d_bucketNodes.data()),
        thrust::raw_pointer_cast(d_groundCubes.data()), thrust::raw_pointer_cast(d_bucketCubes.data()));
    hipDeviceSynchronize();
  };

  void updateMessage() {
    auto time = this->get_clock()->now();

    geometry_msgs::msg::Point p0, p1, p2;

    thrust::copy(d_groundCubes.cbegin(), d_groundCubes.cend(), h_groundCubes.begin());
    hipDeviceSynchronize();
    m_markerArray.markers.at(1).points.clear();
    m_markerArray.markers.at(1).header.stamp = time;
    for (uint32_t i = 0; i < m_cubeCount.x; ++i)
      for (uint32_t j = 0; j < m_cubeCount.y; ++j)
        for (uint32_t k = 0; k < m_cubeCount.z; ++k) {
          const uint32_t c_idx = i * m_cubeCount.y * m_cubeCount.z + j * m_cubeCount.z + k;
          const uint8_t c_type = h_groundCubes[c_idx];
          uint32_t t_idx = c_type * 16;
          int8_t p = m_sampleTriangles[t_idx];
          while (p >= 0) {
            p0.x = (float(i) + m_points.at(p * 3 + 0)) * m_size;
            p0.y = (float(j) + m_points.at(p * 3 + 1)) * m_size;
            p0.z = (float(k) + m_points.at(p * 3 + 2)) * m_size;
            ++t_idx;
            p = m_sampleTriangles[t_idx];
            p1.x = (float(i) + m_points.at(p * 3 + 0)) * m_size;
            p1.y = (float(j) + m_points.at(p * 3 + 1)) * m_size;
            p1.z = (float(k) + m_points.at(p * 3 + 2)) * m_size;
            ++t_idx;
            p = m_sampleTriangles[t_idx];
            p2.x = (float(i) + m_points.at(p * 3 + 0)) * m_size;
            p2.y = (float(j) + m_points.at(p * 3 + 1)) * m_size;
            p2.z = (float(k) + m_points.at(p * 3 + 2)) * m_size;
            m_markerArray.markers.at(1).points.push_back(p2);
            m_markerArray.markers.at(1).points.push_back(p1);
            m_markerArray.markers.at(1).points.push_back(p0);

            ++t_idx;
            p = m_sampleTriangles[t_idx];
          }
        }

    thrust::copy(d_bucketCubes.cbegin(), d_bucketCubes.cend(), h_bucketCubes.begin());
    hipDeviceSynchronize();
    m_markerArray.markers.at(2).points.clear();
    m_markerArray.markers.at(2).header.stamp = time;
    for (uint32_t i = 0; i < m_cubeCount.x; ++i)
      for (uint32_t j = 0; j < m_cubeCount.y; ++j)
        for (uint32_t k = 0; k < m_cubeCount.z; ++k) {
          const uint32_t c_idx = i * m_cubeCount.y * m_cubeCount.z + j * m_cubeCount.z + k;
          const uint8_t c_type = h_bucketCubes[c_idx];
          uint32_t t_idx = c_type * 16;
          int8_t p = m_sampleTriangles[t_idx];
          while (p >= 0) {
            p0.x = (float(i) + m_points.at(p * 3 + 0)) * m_size;
            p0.y = (float(j) + m_points.at(p * 3 + 1)) * m_size;
            p0.z = (float(k) + m_points.at(p * 3 + 2)) * m_size;
            ++t_idx;
            p = m_sampleTriangles[t_idx];
            p1.x = (float(i) + m_points.at(p * 3 + 0)) * m_size;
            p1.y = (float(j) + m_points.at(p * 3 + 1)) * m_size;
            p1.z = (float(k) + m_points.at(p * 3 + 2)) * m_size;
            ++t_idx;
            p = m_sampleTriangles[t_idx];
            p2.x = (float(i) + m_points.at(p * 3 + 0)) * m_size;
            p2.y = (float(j) + m_points.at(p * 3 + 1)) * m_size;
            p2.z = (float(k) + m_points.at(p * 3 + 2)) * m_size;
            m_markerArray.markers.at(2).points.push_back(p2);
            m_markerArray.markers.at(2).points.push_back(p1);
            m_markerArray.markers.at(2).points.push_back(p0);

            ++t_idx;
            p = m_sampleTriangles[t_idx];
          }
        }

    m_markerPublisher->publish(m_markerArray);
  };

  void setSize(const float size) { m_size = size; };

  inline uint32_t cube_ijk2idx(const uint32_t i, const uint32_t j, const uint32_t k) const {
    return i * m_cubeCount.y * m_cubeCount.z + j * m_cubeCount.z + k;
  };

  inline uint32_t node_ijk2idx(const uint32_t i, const uint32_t j, const uint32_t k) const {
    return i * m_nodeCount.y * m_cubeCount.z + j * m_cubeCount.z + k;
  };

  void cube_idx2ijk(const uint32_t idx, Vec3<uint32_t>* out) const {
    out->z = idx % m_cubeCount.z;
    out->y = ((idx - out->z) / m_cubeCount.z) % m_cubeCount.y;
    out->x = (idx - out->z - out->y * m_cubeCount.y) % (m_cubeCount.y * m_cubeCount.z);
  };

  void node_idx2ijk(const uint32_t idx, Vec3<uint32_t>* out) const {
    out->z = idx % m_nodeCount.z;
    out->y = ((idx - out->z) / m_nodeCount.z) % m_nodeCount.y;
    out->x = (idx - out->z - out->y * m_nodeCount.y) % (m_nodeCount.y * m_nodeCount.z);
  };

 private:
  std::shared_ptr<tf2_ros::TransformListener> m_tfListener{nullptr};
  std::unique_ptr<tf2_ros::Buffer> m_tfBuffer;

  double m_startTime;
  float m_size = 1.0f;
  Vec3<uint32_t> m_cubeCount = Vec3<uint32_t>(0, 0, 0);
  Vec3<uint32_t> m_nodeCount = Vec3<uint32_t>(0, 0, 0);
  thrust::device_vector<uint8_t> d_groundCubes;
  thrust::device_vector<uint8_t> d_bucketCubes;
  thrust::host_vector<uint8_t> h_groundCubes;
  thrust::host_vector<uint8_t> h_bucketCubes;
  thrust::device_vector<bool> d_groundNodes;
  thrust::device_vector<bool> d_bucketNodes;
  thrust::device_vector<float> d_groundHeights;
  thrust::device_vector<float> d_groundVolumes;
  thrust::device_vector<float> d_bucketVolumes;
  float m_totalGroundVolume = 0.0f;
  float m_totalBucketVolume = 0.0f;
  visualization_msgs::msg::MarkerArray m_markerArray;
  rclcpp::Subscription<visualization_msgs::msg::MarkerArray>::SharedPtr m_bucketSubsciber;
  rclcpp::Publisher<visualization_msgs::msg::MarkerArray>::SharedPtr m_markerPublisher;

  int32_t count = 100;

  const thrust::device_vector<float> d_sampleVolumes = {
      0.00000000, 0.02083333, 0.02083333, 0.12500000, 0.02083333, 0.04166667, 0.12500000, 0.35416667, 0.02083333, 0.12500000, 0.04166667, 0.35416667,
      0.12500000, 0.35416667, 0.35416667, 0.50000000, 0.02083333, 0.12500000, 0.04166667, 0.35416667, 0.04166667, 0.14583333, 0.14583333, 0.50000000,
      0.04166667, 0.35416667, 0.06250000, 0.50000000, 0.14583333, 0.50000000, 0.37500000, 0.64583333, 0.02083333, 0.04166667, 0.12500000, 0.35416667,
      0.04166667, 0.06250000, 0.35416667, 0.50000000, 0.04166667, 0.14583333, 0.14583333, 0.50000000, 0.14583333, 0.37500000, 0.50000000, 0.64583333,
      0.12500000, 0.35416667, 0.35416667, 0.50000000, 0.14583333, 0.37500000, 0.50000000, 0.64583333, 0.14583333, 0.50000000, 0.37500000, 0.64583333,
      0.25000000, 0.85416667, 0.85416667, 0.87500000, 0.02083333, 0.04166667, 0.04166667, 0.14583333, 0.12500000, 0.14583333, 0.35416667, 0.50000000,
      0.04166667, 0.14583333, 0.06250000, 0.37500000, 0.35416667, 0.50000000, 0.50000000, 0.64583333, 0.04166667, 0.14583333, 0.06250000, 0.37500000,
      0.14583333, 0.25000000, 0.37500000, 0.85416667, 0.06250000, 0.37500000, 0.08333333, 0.93750000, 0.37500000, 0.85416667, 0.93750000, 0.95833333,
      0.12500000, 0.14583333, 0.35416667, 0.50000000, 0.35416667, 0.37500000, 0.50000000, 0.64583333, 0.14583333, 0.25000000, 0.37500000, 0.85416667,
      0.50000000, 0.85416667, 0.64583333, 0.87500000, 0.35416667, 0.50000000, 0.50000000, 0.64583333, 0.50000000, 0.85416667, 0.64583333, 0.87500000,
      0.37500000, 0.85416667, 0.93750000, 0.95833333, 0.85416667, 0.95833333, 0.95833333, 0.97916667, 0.02083333, 0.04166667, 0.04166667, 0.14583333,
      0.04166667, 0.06250000, 0.14583333, 0.37500000, 0.12500000, 0.35416667, 0.14583333, 0.50000000, 0.35416667, 0.50000000, 0.50000000, 0.64583333,
      0.12500000, 0.35416667, 0.14583333, 0.50000000, 0.14583333, 0.37500000, 0.25000000, 0.85416667, 0.35416667, 0.50000000, 0.37500000, 0.64583333,
      0.50000000, 0.64583333, 0.85416667, 0.87500000, 0.04166667, 0.06250000, 0.14583333, 0.37500000, 0.06250000, 0.08333333, 0.37500000, 0.93750000,
      0.14583333, 0.37500000, 0.25000000, 0.85416667, 0.37500000, 0.93750000, 0.85416667, 0.95833333, 0.35416667, 0.50000000, 0.50000000, 0.64583333,
      0.37500000, 0.93750000, 0.85416667, 0.95833333, 0.50000000, 0.64583333, 0.85416667, 0.87500000, 0.85416667, 0.95833333, 0.95833333, 0.97916667,
      0.12500000, 0.14583333, 0.14583333, 0.25000000, 0.35416667, 0.37500000, 0.50000000, 0.85416667, 0.35416667, 0.50000000, 0.37500000, 0.85416667,
      0.50000000, 0.64583333, 0.64583333, 0.87500000, 0.35416667, 0.50000000, 0.37500000, 0.85416667, 0.50000000, 0.85416667, 0.85416667, 0.95833333,
      0.50000000, 0.64583333, 0.93750000, 0.95833333, 0.64583333, 0.87500000, 0.95833333, 0.97916667, 0.35416667, 0.37500000, 0.50000000, 0.85416667,
      0.50000000, 0.93750000, 0.64583333, 0.95833333, 0.50000000, 0.85416667, 0.85416667, 0.95833333, 0.64583333, 0.95833333, 0.87500000, 0.97916667,
      0.50000000, 0.64583333, 0.64583333, 0.87500000, 0.64583333, 0.95833333, 0.87500000, 0.97916667, 0.64583333, 0.87500000, 0.95833333, 0.97916667,
      0.87500000, 0.97916667, 0.97916667, 1.00000000};

  const std::array<float, 12 * 3> m_points = {0.5f, 0.0f, 0.0f, 1.0f, 0.5f, 0.0f, 0.5f, 1.0f, 0.0f, 0.0f, 0.5f, 0.0f,
                                              0.5f, 0.0f, 1.0f, 1.0f, 0.5f, 1.0f, 0.5f, 1.0f, 1.0f, 0.0f, 0.5f, 1.0f,
                                              0.0f, 0.0f, 0.5f, 1.0f, 0.0f, 0.5f, 1.0f, 1.0f, 0.5f, 0.0f, 1.0f, 0.5f};

  const std::array<int8_t, 256 * 16> m_sampleTriangles = {
      -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0,  8,  3,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0,  1,  9,  -1,
      -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1,  8,  3,  9,  8,  1,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1,  2,  10, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, -1, -1, -1, -1, 0,  8,  3,  1,  2,  10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 9,  2,  10, 0,  2,  9,  -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 2,  8,  3,  2,  10, 8,  10, 9,  8,  -1, -1, -1, -1, -1, -1, -1, 3,  11, 2,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
      0,  11, 2,  8,  11, 0,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1,  9,  0,  2,  3,  11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1,  11, 2,  1,
      9,  11, 9,  8,  11, -1, -1, -1, -1, -1, -1, -1, 3,  10, 1,  11, 10, 3,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0,  10, 1,  0,  8,  10, 8,  11,
      10, -1, -1, -1, -1, -1, -1, -1, 3,  9,  0,  3,  11, 9,  11, 10, 9,  -1, -1, -1, -1, -1, -1, -1, 9,  8,  10, 10, 8,  11, -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 4,  7,  8,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 4,  3,  0,  7,  3,  4,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
      0,  1,  9,  8,  4,  7,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 4,  1,  9,  4,  7,  1,  7,  3,  1,  -1, -1, -1, -1, -1, -1, -1, 1,  2,  10, 8,
      4,  7,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 3,  4,  7,  3,  0,  4,  1,  2,  10, -1, -1, -1, -1, -1, -1, -1, 9,  2,  10, 9,  0,  2,  8,  4,
      7,  -1, -1, -1, -1, -1, -1, -1, 2,  10, 9,  2,  9,  7,  2,  7,  3,  7,  9,  4,  -1, -1, -1, -1, 8,  4,  7,  3,  11, 2,  -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 11, 4,  7,  11, 2,  4,  2,  0,  4,  -1, -1, -1, -1, -1, -1, -1, 9,  0,  1,  8,  4,  7,  2,  3,  11, -1, -1, -1, -1, -1, -1, -1,
      4,  7,  11, 9,  4,  11, 9,  11, 2,  9,  2,  1,  -1, -1, -1, -1, 3,  10, 1,  3,  11, 10, 7,  8,  4,  -1, -1, -1, -1, -1, -1, -1, 1,  11, 10, 1,
      4,  11, 1,  0,  4,  7,  11, 4,  -1, -1, -1, -1, 4,  7,  8,  9,  0,  11, 9,  11, 10, 11, 0,  3,  -1, -1, -1, -1, 4,  7,  11, 4,  11, 9,  9,  11,
      10, -1, -1, -1, -1, -1, -1, -1, 9,  5,  4,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 9,  5,  4,  0,  8,  3,  -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 0,  5,  4,  1,  5,  0,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 8,  5,  4,  8,  3,  5,  3,  1,  5,  -1, -1, -1, -1, -1, -1, -1,
      1,  2,  10, 9,  5,  4,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 3,  0,  8,  1,  2,  10, 4,  9,  5,  -1, -1, -1, -1, -1, -1, -1, 5,  2,  10, 5,
      4,  2,  4,  0,  2,  -1, -1, -1, -1, -1, -1, -1, 2,  10, 5,  3,  2,  5,  3,  5,  4,  3,  4,  8,  -1, -1, -1, -1, 9,  5,  4,  2,  3,  11, -1, -1,
      -1, -1, -1, -1, -1, -1, -1, -1, 0,  11, 2,  0,  8,  11, 4,  9,  5,  -1, -1, -1, -1, -1, -1, -1, 0,  5,  4,  0,  1,  5,  2,  3,  11, -1, -1, -1,
      -1, -1, -1, -1, 2,  1,  5,  2,  5,  8,  2,  8,  11, 4,  8,  5,  -1, -1, -1, -1, 10, 3,  11, 10, 1,  3,  9,  5,  4,  -1, -1, -1, -1, -1, -1, -1,
      4,  9,  5,  0,  8,  1,  8,  10, 1,  8,  11, 10, -1, -1, -1, -1, 5,  4,  0,  5,  0,  11, 5,  11, 10, 11, 0,  3,  -1, -1, -1, -1, 5,  4,  8,  5,
      8,  10, 10, 8,  11, -1, -1, -1, -1, -1, -1, -1, 9,  7,  8,  5,  7,  9,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 9,  3,  0,  9,  5,  3,  5,  7,
      3,  -1, -1, -1, -1, -1, -1, -1, 0,  7,  8,  0,  1,  7,  1,  5,  7,  -1, -1, -1, -1, -1, -1, -1, 1,  5,  3,  3,  5,  7,  -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 9,  7,  8,  9,  5,  7,  10, 1,  2,  -1, -1, -1, -1, -1, -1, -1, 10, 1,  2,  9,  5,  0,  5,  3,  0,  5,  7,  3,  -1, -1, -1, -1,
      8,  0,  2,  8,  2,  5,  8,  5,  7,  10, 5,  2,  -1, -1, -1, -1, 2,  10, 5,  2,  5,  3,  3,  5,  7,  -1, -1, -1, -1, -1, -1, -1, 7,  9,  5,  7,
      8,  9,  3,  11, 2,  -1, -1, -1, -1, -1, -1, -1, 9,  5,  7,  9,  7,  2,  9,  2,  0,  2,  7,  11, -1, -1, -1, -1, 2,  3,  11, 0,  1,  8,  1,  7,
      8,  1,  5,  7,  -1, -1, -1, -1, 11, 2,  1,  11, 1,  7,  7,  1,  5,  -1, -1, -1, -1, -1, -1, -1, 9,  5,  8,  8,  5,  7,  10, 1,  3,  10, 3,  11,
      -1, -1, -1, -1, 5,  7,  0,  5,  0,  9,  7,  11, 0,  1,  0,  10, 11, 10, 0,  -1, 11, 10, 0,  11, 0,  3,  10, 5,  0,  8,  0,  7,  5,  7,  0,  -1,
      11, 10, 5,  7,  11, 5,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 10, 6,  5,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0,  8,  3,  5,
      10, 6,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 9,  0,  1,  5,  10, 6,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1,  8,  3,  1,  9,  8,  5,  10,
      6,  -1, -1, -1, -1, -1, -1, -1, 1,  6,  5,  2,  6,  1,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1,  6,  5,  1,  2,  6,  3,  0,  8,  -1, -1, -1,
      -1, -1, -1, -1, 9,  6,  5,  9,  0,  6,  0,  2,  6,  -1, -1, -1, -1, -1, -1, -1, 5,  9,  8,  5,  8,  2,  5,  2,  6,  3,  2,  8,  -1, -1, -1, -1,
      2,  3,  11, 10, 6,  5,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 11, 0,  8,  11, 2,  0,  10, 6,  5,  -1, -1, -1, -1, -1, -1, -1, 0,  1,  9,  2,
      3,  11, 5,  10, 6,  -1, -1, -1, -1, -1, -1, -1, 5,  10, 6,  1,  9,  2,  9,  11, 2,  9,  8,  11, -1, -1, -1, -1, 6,  3,  11, 6,  5,  3,  5,  1,
      3,  -1, -1, -1, -1, -1, -1, -1, 0,  8,  11, 0,  11, 5,  0,  5,  1,  5,  11, 6,  -1, -1, -1, -1, 3,  11, 6,  0,  3,  6,  0,  6,  5,  0,  5,  9,
      -1, -1, -1, -1, 6,  5,  9,  6,  9,  11, 11, 9,  8,  -1, -1, -1, -1, -1, -1, -1, 5,  10, 6,  4,  7,  8,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
      4,  3,  0,  4,  7,  3,  6,  5,  10, -1, -1, -1, -1, -1, -1, -1, 1,  9,  0,  5,  10, 6,  8,  4,  7,  -1, -1, -1, -1, -1, -1, -1, 10, 6,  5,  1,
      9,  7,  1,  7,  3,  7,  9,  4,  -1, -1, -1, -1, 6,  1,  2,  6,  5,  1,  4,  7,  8,  -1, -1, -1, -1, -1, -1, -1, 1,  2,  5,  5,  2,  6,  3,  0,
      4,  3,  4,  7,  -1, -1, -1, -1, 8,  4,  7,  9,  0,  5,  0,  6,  5,  0,  2,  6,  -1, -1, -1, -1, 7,  3,  9,  7,  9,  4,  3,  2,  9,  5,  9,  6,
      2,  6,  9,  -1, 3,  11, 2,  7,  8,  4,  10, 6,  5,  -1, -1, -1, -1, -1, -1, -1, 5,  10, 6,  4,  7,  2,  4,  2,  0,  2,  7,  11, -1, -1, -1, -1,
      0,  1,  9,  4,  7,  8,  2,  3,  11, 5,  10, 6,  -1, -1, -1, -1, 9,  2,  1,  9,  11, 2,  9,  4,  11, 7,  11, 4,  5,  10, 6,  -1, 8,  4,  7,  3,
      11, 5,  3,  5,  1,  5,  11, 6,  -1, -1, -1, -1, 5,  1,  11, 5,  11, 6,  1,  0,  11, 7,  11, 4,  0,  4,  11, -1, 0,  5,  9,  0,  6,  5,  0,  3,
      6,  11, 6,  3,  8,  4,  7,  -1, 6,  5,  9,  6,  9,  11, 4,  7,  9,  7,  11, 9,  -1, -1, -1, -1, 10, 4,  9,  6,  4,  10, -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 4,  10, 6,  4,  9,  10, 0,  8,  3,  -1, -1, -1, -1, -1, -1, -1, 10, 0,  1,  10, 6,  0,  6,  4,  0,  -1, -1, -1, -1, -1, -1, -1,
      8,  3,  1,  8,  1,  6,  8,  6,  4,  6,  1,  10, -1, -1, -1, -1, 1,  4,  9,  1,  2,  4,  2,  6,  4,  -1, -1, -1, -1, -1, -1, -1, 3,  0,  8,  1,
      2,  9,  2,  4,  9,  2,  6,  4,  -1, -1, -1, -1, 0,  2,  4,  4,  2,  6,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 8,  3,  2,  8,  2,  4,  4,  2,
      6,  -1, -1, -1, -1, -1, -1, -1, 10, 4,  9,  10, 6,  4,  11, 2,  3,  -1, -1, -1, -1, -1, -1, -1, 0,  8,  2,  2,  8,  11, 4,  9,  10, 4,  10, 6,
      -1, -1, -1, -1, 3,  11, 2,  0,  1,  6,  0,  6,  4,  6,  1,  10, -1, -1, -1, -1, 6,  4,  1,  6,  1,  10, 4,  8,  1,  2,  1,  11, 8,  11, 1,  -1,
      9,  6,  4,  9,  3,  6,  9,  1,  3,  11, 6,  3,  -1, -1, -1, -1, 8,  11, 1,  8,  1,  0,  11, 6,  1,  9,  1,  4,  6,  4,  1,  -1, 3,  11, 6,  3,
      6,  0,  0,  6,  4,  -1, -1, -1, -1, -1, -1, -1, 6,  4,  8,  11, 6,  8,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 7,  10, 6,  7,  8,  10, 8,  9,
      10, -1, -1, -1, -1, -1, -1, -1, 0,  7,  3,  0,  10, 7,  0,  9,  10, 6,  7,  10, -1, -1, -1, -1, 10, 6,  7,  1,  10, 7,  1,  7,  8,  1,  8,  0,
      -1, -1, -1, -1, 10, 6,  7,  10, 7,  1,  1,  7,  3,  -1, -1, -1, -1, -1, -1, -1, 1,  2,  6,  1,  6,  8,  1,  8,  9,  8,  6,  7,  -1, -1, -1, -1,
      2,  6,  9,  2,  9,  1,  6,  7,  9,  0,  9,  3,  7,  3,  9,  -1, 7,  8,  0,  7,  0,  6,  6,  0,  2,  -1, -1, -1, -1, -1, -1, -1, 7,  3,  2,  6,
      7,  2,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 2,  3,  11, 10, 6,  8,  10, 8,  9,  8,  6,  7,  -1, -1, -1, -1, 2,  0,  7,  2,  7,  11, 0,  9,
      7,  6,  7,  10, 9,  10, 7,  -1, 1,  8,  0,  1,  7,  8,  1,  10, 7,  6,  7,  10, 2,  3,  11, -1, 11, 2,  1,  11, 1,  7,  10, 6,  1,  6,  7,  1,
      -1, -1, -1, -1, 8,  9,  6,  8,  6,  7,  9,  1,  6,  11, 6,  3,  1,  3,  6,  -1, 0,  9,  1,  11, 6,  7,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
      7,  8,  0,  7,  0,  6,  3,  11, 0,  11, 6,  0,  -1, -1, -1, -1, 7,  11, 6,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 7,  6,  11, -1,
      -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 3,  0,  8,  11, 7,  6,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0,  1,  9,  11, 7,  6,  -1, -1,
      -1, -1, -1, -1, -1, -1, -1, -1, 8,  1,  9,  8,  3,  1,  11, 7,  6,  -1, -1, -1, -1, -1, -1, -1, 10, 1,  2,  6,  11, 7,  -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 1,  2,  10, 3,  0,  8,  6,  11, 7,  -1, -1, -1, -1, -1, -1, -1, 2,  9,  0,  2,  10, 9,  6,  11, 7,  -1, -1, -1, -1, -1, -1, -1,
      6,  11, 7,  2,  10, 3,  10, 8,  3,  10, 9,  8,  -1, -1, -1, -1, 7,  2,  3,  6,  2,  7,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 7,  0,  8,  7,
      6,  0,  6,  2,  0,  -1, -1, -1, -1, -1, -1, -1, 2,  7,  6,  2,  3,  7,  0,  1,  9,  -1, -1, -1, -1, -1, -1, -1, 1,  6,  2,  1,  8,  6,  1,  9,
      8,  8,  7,  6,  -1, -1, -1, -1, 10, 7,  6,  10, 1,  7,  1,  3,  7,  -1, -1, -1, -1, -1, -1, -1, 10, 7,  6,  1,  7,  10, 1,  8,  7,  1,  0,  8,
      -1, -1, -1, -1, 0,  3,  7,  0,  7,  10, 0,  10, 9,  6,  10, 7,  -1, -1, -1, -1, 7,  6,  10, 7,  10, 8,  8,  10, 9,  -1, -1, -1, -1, -1, -1, -1,
      6,  8,  4,  11, 8,  6,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 3,  6,  11, 3,  0,  6,  0,  4,  6,  -1, -1, -1, -1, -1, -1, -1, 8,  6,  11, 8,
      4,  6,  9,  0,  1,  -1, -1, -1, -1, -1, -1, -1, 9,  4,  6,  9,  6,  3,  9,  3,  1,  11, 3,  6,  -1, -1, -1, -1, 6,  8,  4,  6,  11, 8,  2,  10,
      1,  -1, -1, -1, -1, -1, -1, -1, 1,  2,  10, 3,  0,  11, 0,  6,  11, 0,  4,  6,  -1, -1, -1, -1, 4,  11, 8,  4,  6,  11, 0,  2,  9,  2,  10, 9,
      -1, -1, -1, -1, 10, 9,  3,  10, 3,  2,  9,  4,  3,  11, 3,  6,  4,  6,  3,  -1, 8,  2,  3,  8,  4,  2,  4,  6,  2,  -1, -1, -1, -1, -1, -1, -1,
      0,  4,  2,  4,  6,  2,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1,  9,  0,  2,  3,  4,  2,  4,  6,  4,  3,  8,  -1, -1, -1, -1, 1,  9,  4,  1,
      4,  2,  2,  4,  6,  -1, -1, -1, -1, -1, -1, -1, 8,  1,  3,  8,  6,  1,  8,  4,  6,  6,  10, 1,  -1, -1, -1, -1, 10, 1,  0,  10, 0,  6,  6,  0,
      4,  -1, -1, -1, -1, -1, -1, -1, 4,  6,  3,  4,  3,  8,  6,  10, 3,  0,  3,  9,  10, 9,  3,  -1, 10, 9,  4,  6,  10, 4,  -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 4,  9,  5,  7,  6,  11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0,  8,  3,  4,  9,  5,  11, 7,  6,  -1, -1, -1, -1, -1, -1, -1,
      5,  0,  1,  5,  4,  0,  7,  6,  11, -1, -1, -1, -1, -1, -1, -1, 11, 7,  6,  8,  3,  4,  3,  5,  4,  3,  1,  5,  -1, -1, -1, -1, 9,  5,  4,  10,
      1,  2,  7,  6,  11, -1, -1, -1, -1, -1, -1, -1, 6,  11, 7,  1,  2,  10, 0,  8,  3,  4,  9,  5,  -1, -1, -1, -1, 7,  6,  11, 5,  4,  10, 4,  2,
      10, 4,  0,  2,  -1, -1, -1, -1, 3,  4,  8,  3,  5,  4,  3,  2,  5,  10, 5,  2,  11, 7,  6,  -1, 7,  2,  3,  7,  6,  2,  5,  4,  9,  -1, -1, -1,
      -1, -1, -1, -1, 9,  5,  4,  0,  8,  6,  0,  6,  2,  6,  8,  7,  -1, -1, -1, -1, 3,  6,  2,  3,  7,  6,  1,  5,  0,  5,  4,  0,  -1, -1, -1, -1,
      6,  2,  8,  6,  8,  7,  2,  1,  8,  4,  8,  5,  1,  5,  8,  -1, 9,  5,  4,  10, 1,  6,  1,  7,  6,  1,  3,  7,  -1, -1, -1, -1, 1,  6,  10, 1,
      7,  6,  1,  0,  7,  8,  7,  0,  9,  5,  4,  -1, 4,  0,  10, 4,  10, 5,  0,  3,  10, 6,  10, 7,  3,  7,  10, -1, 7,  6,  10, 7,  10, 8,  5,  4,
      10, 4,  8,  10, -1, -1, -1, -1, 6,  9,  5,  6,  11, 9,  11, 8,  9,  -1, -1, -1, -1, -1, -1, -1, 3,  6,  11, 0,  6,  3,  0,  5,  6,  0,  9,  5,
      -1, -1, -1, -1, 0,  11, 8,  0,  5,  11, 0,  1,  5,  5,  6,  11, -1, -1, -1, -1, 6,  11, 3,  6,  3,  5,  5,  3,  1,  -1, -1, -1, -1, -1, -1, -1,
      1,  2,  10, 9,  5,  11, 9,  11, 8,  11, 5,  6,  -1, -1, -1, -1, 0,  11, 3,  0,  6,  11, 0,  9,  6,  5,  6,  9,  1,  2,  10, -1, 11, 8,  5,  11,
      5,  6,  8,  0,  5,  10, 5,  2,  0,  2,  5,  -1, 6,  11, 3,  6,  3,  5,  2,  10, 3,  10, 5,  3,  -1, -1, -1, -1, 5,  8,  9,  5,  2,  8,  5,  6,
      2,  3,  8,  2,  -1, -1, -1, -1, 9,  5,  6,  9,  6,  0,  0,  6,  2,  -1, -1, -1, -1, -1, -1, -1, 1,  5,  8,  1,  8,  0,  5,  6,  8,  3,  8,  2,
      6,  2,  8,  -1, 1,  5,  6,  2,  1,  6,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1,  3,  6,  1,  6,  10, 3,  8,  6,  5,  6,  9,  8,  9,  6,  -1,
      10, 1,  0,  10, 0,  6,  9,  5,  0,  5,  6,  0,  -1, -1, -1, -1, 0,  3,  8,  5,  6,  10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 10, 5,  6,  -1,
      -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 11, 5,  10, 7,  5,  11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 11, 5,  10, 11, 7,  5,  8,  3,
      0,  -1, -1, -1, -1, -1, -1, -1, 5,  11, 7,  5,  10, 11, 1,  9,  0,  -1, -1, -1, -1, -1, -1, -1, 10, 7,  5,  10, 11, 7,  9,  8,  1,  8,  3,  1,
      -1, -1, -1, -1, 11, 1,  2,  11, 7,  1,  7,  5,  1,  -1, -1, -1, -1, -1, -1, -1, 0,  8,  3,  1,  2,  7,  1,  7,  5,  7,  2,  11, -1, -1, -1, -1,
      9,  7,  5,  9,  2,  7,  9,  0,  2,  2,  11, 7,  -1, -1, -1, -1, 7,  5,  2,  7,  2,  11, 5,  9,  2,  3,  2,  8,  9,  8,  2,  -1, 2,  5,  10, 2,
      3,  5,  3,  7,  5,  -1, -1, -1, -1, -1, -1, -1, 8,  2,  0,  8,  5,  2,  8,  7,  5,  10, 2,  5,  -1, -1, -1, -1, 9,  0,  1,  5,  10, 3,  5,  3,
      7,  3,  10, 2,  -1, -1, -1, -1, 9,  8,  2,  9,  2,  1,  8,  7,  2,  10, 2,  5,  7,  5,  2,  -1, 1,  3,  5,  3,  7,  5,  -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 0,  8,  7,  0,  7,  1,  1,  7,  5,  -1, -1, -1, -1, -1, -1, -1, 9,  0,  3,  9,  3,  5,  5,  3,  7,  -1, -1, -1, -1, -1, -1, -1,
      9,  8,  7,  5,  9,  7,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 5,  8,  4,  5,  10, 8,  10, 11, 8,  -1, -1, -1, -1, -1, -1, -1, 5,  0,  4,  5,
      11, 0,  5,  10, 11, 11, 3,  0,  -1, -1, -1, -1, 0,  1,  9,  8,  4,  10, 8,  10, 11, 10, 4,  5,  -1, -1, -1, -1, 10, 11, 4,  10, 4,  5,  11, 3,
      4,  9,  4,  1,  3,  1,  4,  -1, 2,  5,  1,  2,  8,  5,  2,  11, 8,  4,  5,  8,  -1, -1, -1, -1, 0,  4,  11, 0,  11, 3,  4,  5,  11, 2,  11, 1,
      5,  1,  11, -1, 0,  2,  5,  0,  5,  9,  2,  11, 5,  4,  5,  8,  11, 8,  5,  -1, 9,  4,  5,  2,  11, 3,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
      2,  5,  10, 3,  5,  2,  3,  4,  5,  3,  8,  4,  -1, -1, -1, -1, 5,  10, 2,  5,  2,  4,  4,  2,  0,  -1, -1, -1, -1, -1, -1, -1, 3,  10, 2,  3,
      5,  10, 3,  8,  5,  4,  5,  8,  0,  1,  9,  -1, 5,  10, 2,  5,  2,  4,  1,  9,  2,  9,  4,  2,  -1, -1, -1, -1, 8,  4,  5,  8,  5,  3,  3,  5,
      1,  -1, -1, -1, -1, -1, -1, -1, 0,  4,  5,  1,  0,  5,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 8,  4,  5,  8,  5,  3,  9,  0,  5,  0,  3,  5,
      -1, -1, -1, -1, 9,  4,  5,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 4,  11, 7,  4,  9,  11, 9,  10, 11, -1, -1, -1, -1, -1, -1, -1,
      0,  8,  3,  4,  9,  7,  9,  11, 7,  9,  10, 11, -1, -1, -1, -1, 1,  10, 11, 1,  11, 4,  1,  4,  0,  7,  4,  11, -1, -1, -1, -1, 3,  1,  4,  3,
      4,  8,  1,  10, 4,  7,  4,  11, 10, 11, 4,  -1, 4,  11, 7,  9,  11, 4,  9,  2,  11, 9,  1,  2,  -1, -1, -1, -1, 9,  7,  4,  9,  11, 7,  9,  1,
      11, 2,  11, 1,  0,  8,  3,  -1, 11, 7,  4,  11, 4,  2,  2,  4,  0,  -1, -1, -1, -1, -1, -1, -1, 11, 7,  4,  11, 4,  2,  8,  3,  4,  3,  2,  4,
      -1, -1, -1, -1, 2,  9,  10, 2,  7,  9,  2,  3,  7,  7,  4,  9,  -1, -1, -1, -1, 9,  10, 7,  9,  7,  4,  10, 2,  7,  8,  7,  0,  2,  0,  7,  -1,
      3,  7,  10, 3,  10, 2,  7,  4,  10, 1,  10, 0,  4,  0,  10, -1, 1,  10, 2,  8,  7,  4,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 4,  9,  1,  4,
      1,  7,  7,  1,  3,  -1, -1, -1, -1, -1, -1, -1, 4,  9,  1,  4,  1,  7,  0,  8,  1,  8,  7,  1,  -1, -1, -1, -1, 4,  0,  3,  7,  4,  3,  -1, -1,
      -1, -1, -1, -1, -1, -1, -1, -1, 4,  8,  7,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 9,  10, 8,  10, 11, 8,  -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 3,  0,  9,  3,  9,  11, 11, 9,  10, -1, -1, -1, -1, -1, -1, -1, 0,  1,  10, 0,  10, 8,  8,  10, 11, -1, -1, -1, -1, -1, -1, -1,
      3,  1,  10, 11, 3,  10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 1,  2,  11, 1,  11, 9,  9,  11, 8,  -1, -1, -1, -1, -1, -1, -1, 3,  0,  9,  3,
      9,  11, 1,  2,  9,  2,  11, 9,  -1, -1, -1, -1, 0,  2,  11, 8,  0,  11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 3,  2,  11, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, -1, -1, -1, -1, 2,  3,  8,  2,  8,  10, 10, 8,  9,  -1, -1, -1, -1, -1, -1, -1, 9,  10, 2,  0,  9,  2,  -1, -1, -1, -1, -1, -1,
      -1, -1, -1, -1, 2,  3,  8,  2,  8,  10, 0,  1,  8,  1,  10, 8,  -1, -1, -1, -1, 1,  10, 2,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1,
      1,  3,  8,  9,  1,  8,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0,  9,  1,  -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, 0,  3,  8,  -1,
      -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1};
};

int main(int argc, char** argv) {
  rclcpp::init(argc, argv);
  rclcpp::spin(std::make_shared<MarchingCubes>(400, 400, 100, 0.01));
  rclcpp::shutdown();
  return 0;
}
